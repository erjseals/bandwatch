#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <malloc.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h> 
#include <math.h>
#include <getopt.h>
#include <unistd.h>
#include <iostream>
#include <signal.h>
#include <sys/time.h>

#define LOG 1

#define CUDA_MEMCPY 0
#define CUDA_MEMSET 1
#define CUDA_D2D 2
#define CUDA_C_KERNEL 3
#define CUDA_C_KERNEL_UVM 4

#define ITERATIONS_DEFAULT 101
#define MODE_DEFAULT CUDA_MEMCPY
#define KILO 1024
#define DATASIZE_DEFAULT 50

void memsets(const size_t datasize, const bool hasToSynch, const size_t iterations);
void memcpys(const size_t datasize, const bool hasToSynch, const size_t iterations);
void copykernel(const bool isUVM, const size_t datasize, const bool hasToSynch, const size_t iterations);
void d2d(const size_t datasize, const bool hasToSynch, const size_t iterations);

volatile size_t iter = 0;
volatile unsigned int g_start = 0;

struct argsStruct {
	bool verbose;
    bool help;
    bool hasToSynch;
	int32_t mode;
    size_t datasize;
    size_t iterations;
};

#if LOG
unsigned int get_usecs()
{
  struct timeval time;
  gettimeofday(&time, NULL);
  return (time.tv_sec * 1000000 + time.tv_usec);
}

void start(){
  iter = 0;
  g_start = get_usecs();
}
void finish(){
  float dur = get_usecs() - g_start; 
  float dur_in_sec = (float)dur / 1000000; 
  printf("Total iterations: %ld\n",iter);
  printf("elapsed = %.2f sec ( %.0f usec )\n", dur_in_sec, dur);
  float bw = (float)iter * 102400 * 1024 * sizeof(float) / dur_in_sec / 1024 / 1024;
  printf("Memcpy BW = %.2f MB/s\n", bw*2);
  printf("Memset BW = %.2f MB/s\n", bw);
  iter = 10000000;
}

void signal_handler(int sigNo)
{
  switch(sigNo) {
  case SIGUSR1: {
    start();
    break;
  }
  case SIGUSR2: {
    finish();
    break;
  }
  default:
    break;
  }
}
#endif


//no boundary checks to avoid unnecessary "if"s.
__global__ void copyKernelGPU(float *a, float *b){

    const uint32_t gid = threadIdx.x + blockIdx.x * blockDim.x; 
    b[gid] = a[gid];
}

void printHelp(){

    std::cout << "Usage: ./cudainterf [-h or --help] [-v or --verbose] [-s or --synch] [-d or --datasize=<size_t>] [-i or --iterations=<size_t>] " << std::endl <<
    "[-m or --mode=<copyKernel|copyKernelUVM|d2d|memset|memcpy>]" << std::endl;
    std::cout << "--help    Display help information" << std::endl;
    std::cout << "--verbose   Self-explanatory. Default is false" << std::endl;
    std::cout << "--synch    Will call hipStreamSynchronize once every two command submssions. Default is false." << std::endl;
    std::cout << "--mode=<copyKernel|copyKernelUVM|d2d|memset|memcpy>   Which interference mode to run. Default is hipMemcpy" << std::endl;
    std::cout << "--iterations=<size_t>	      How many iterations for the innteferring test. Default is " << ITERATIONS_DEFAULT << std::endl;
    std::cout << "--datasize=<size_t>    How many KILO float elements to use in the tests. Default is " << DATASIZE_DEFAULT << " KILO eleements" << std::endl;

    exit(EXIT_SUCCESS);

}

bool parseArgs(argsStruct &args, int argc, char* argv[])
{
    while (1)
    {
        int arg;
        static struct option long_options[] = {{"help", no_argument, 0, 'h'},
            {"datasize", required_argument, 0, 'd'},
            {"verbose", no_argument, 0, 'v'},
            {"synch", no_argument, 0, 's'},
            {"iterations", required_argument, 0, 'i'},
            {"mode", required_argument, 0, 'm'},
            {nullptr, 0, nullptr, 0}};
        int option_index = 0;
        arg = getopt_long(argc, argv, "hvsi:m:d:", long_options, &option_index);
	    if (arg == -1)
        {
            break;
        }

        switch (arg)
        {
        case 'h': args.help = true;
	    case 'v': args.verbose = true; break;
        case 's': args.hasToSynch = true; break;
        case 'i':
            if (optarg)
            {
                args.iterations = atol(optarg);
            }
            break;
        case 'd':
        if (optarg)
            {
                args.datasize = atol(optarg);
            }
        break;
        case 'm':
            if(optarg){
                if(strcmp("memcpy",optarg)==0)
                    args.mode = CUDA_MEMCPY;
                else if (strcmp("memset",optarg)==0)
                    args.mode = CUDA_MEMSET;
                else if (strcmp("d2d",optarg)==0)
                    args.mode = CUDA_D2D;
                else if (strcmp("copyKernel",optarg)==0)
                    args.mode = CUDA_C_KERNEL;
                else if (strcmp("copyKernelUVM",optarg)==0)
                    args.mode = CUDA_C_KERNEL_UVM;
                else { std::cout << "Error in test specification" << std::endl; exit(-1); }
            }
            break;
        default: return false;
        }
    }
    return true;
}


int main(int argc, char *argv[]){

    size_t elements;

    argsStruct args;
    args.verbose = false;
    args.help = false;
    args.datasize = DATASIZE_DEFAULT;
    args.hasToSynch = false;
    args.mode = MODE_DEFAULT;
    args.iterations = ITERATIONS_DEFAULT;

    if(argc<=1)
      args.verbose = true;

    parseArgs(args,argc,argv);

    elements = KILO * args.datasize;

    if(args.help)
        printHelp();

    if(args.verbose){
        printf("Executing with the following parameters: \n");
        printf("Num elements = %zu\n", elements);
	printf("Will iterate for %zu times\n", args.iterations);
        if(args.mode==CUDA_C_KERNEL_UVM) printf("Mode is copyKernelUVM\n");
        else if(args.mode==CUDA_D2D) printf("Mode is cuda device to device copy\n");
        else if(args.mode==CUDA_MEMCPY) printf("Mode is cudaMemcpy\n");
        else if(args.mode==CUDA_MEMSET) printf("Mode is cudaMemset\n");
        else  printf("Mode is copyKernel with no UVM\n");
        printf("Will synch after each couple of commands? %s\n", (args.hasToSynch) ? "true" : "false" );
        fflush(stdout); 
    }

#if LOG
    /* set signals to terminate once time has been reached */
    if (signal(SIGUSR1,signal_handler) == SIG_ERR)
      printf("Failed to setup SIGUSR1\n");
    if (signal(SIGUSR2,signal_handler) == SIG_ERR)
      printf("Failed to setup SIGUSR2\n");
#endif

    const size_t datasize = sizeof(float) * elements;
    const bool hasToSynch = args.hasToSynch;
    const size_t iterations = args.iterations;

    switch(args.mode){
        case CUDA_C_KERNEL_UVM:
            copykernel(true, datasize, hasToSynch, iterations);
        break;
        case CUDA_MEMSET:
            memsets(datasize, hasToSynch, iterations);
        break;
        case CUDA_MEMCPY:
            memcpys(datasize, hasToSynch, iterations);
        break;
        case CUDA_D2D:
            d2d(datasize, hasToSynch, iterations);
        break;
        default: 
            copykernel(false, datasize, hasToSynch, iterations);
    }

    if(args.verbose)
        std::cout << argv[0] << ": Done" << std::endl;

    return EXIT_SUCCESS;
}

void copykernel(const bool isUVM, const size_t datasize, const bool hasToSynch, const size_t iterations){

    hipStream_t s;
    hipStreamCreate(&s);

    float *hData; 
    float *dData0; 
    float *dData1;

    if(!isUVM){
        hipHostMalloc((void**)&hData, datasize, hipHostMallocDefault);
        hipMalloc((void**)&dData0, datasize);
        hipMalloc((void**)&dData1, datasize); 
        memset(hData,101,datasize);
        hipMemcpyAsync(dData0, hData, datasize, hipMemcpyHostToDevice, s);
        hipStreamSynchronize(s);
    }else{
        hipMallocManaged((void**)&dData0,datasize);
        hipMallocManaged((void**)&dData1,datasize);
        memset(dData0,101,datasize);
    }

    const uint32_t threads = 128; //TODO: find best.
    const uint32_t blocks = datasize/sizeof(float)/threads; //TODO: check for remainder

    size_t i = 0;
    while(i<iterations){
        copyKernelGPU<<<blocks,threads,0,s>>>(dData0,dData1);
        copyKernelGPU<<<blocks,threads,0,s>>>(dData1,dData0);
        i++;
        if(hasToSynch) hipStreamSynchronize(s);
    }

    hipFree(dData0);
    hipFree(dData1);
    if(!isUVM)
        hipHostFree(hData);

}

void d2d(const size_t datasize, const bool hasToSynch, const size_t iterations){

    hipStream_t s;
    hipStreamCreate(&s);
    float *hData; hipHostMalloc((void**)&hData, datasize, hipHostMallocDefault);
    float *dData0; hipMalloc((void**)&dData0, datasize);
    float *dData1; hipMalloc((void**)&dData1, datasize); 

    memset(hData,101,datasize);
    hipMemcpyAsync(dData0, hData, datasize, hipMemcpyHostToDevice, s);
    hipStreamSynchronize(s);
    hipHostFree(hData);
    
    size_t i = 0;
    while(i<iterations){
        hipMemcpyAsync(dData0,dData1,datasize, hipMemcpyDeviceToDevice, s);
        hipMemcpyAsync(dData1,dData0,datasize, hipMemcpyDeviceToDevice, s);
        i++;
        if(hasToSynch) hipStreamSynchronize(s);
    }

    hipFree(dData0);
    hipFree(dData1);
}

void memcpys(const size_t datasize, const bool hasToSynch, const size_t iterations){

    hipStream_t s;
    hipStreamCreate(&s);
    float *hData; hipHostMalloc((void**)&hData, datasize, hipHostMallocDefault);
    float *dData; hipMalloc((void**)&dData, datasize);

    memset(hData,101,datasize);
    hipStreamSynchronize(s);

    iter = 0;
    while(iter<iterations){
        hipMemcpyAsync(dData, hData, datasize, hipMemcpyHostToDevice, s);
	      hipMemcpyAsync(hData, dData, datasize, hipMemcpyDeviceToHost, s);
        iter++;
        if(hasToSynch) hipStreamSynchronize(s);
    }

    hipHostFree(hData);
    hipFree(dData);
}

void memsets(const size_t datasize, const bool hasToSynch, const size_t iterations){

    hipStream_t s;
    hipStreamCreate(&s);

    float *hData; hipHostMalloc((void**)&hData, sizeof(float), hipHostMallocDefault); 
    float *dData; hipMalloc((void**)&dData, datasize);
	
    hipStreamSynchronize(s);

    iter = 0;
    while(iter<iterations){
        hipMemset((void**)dData, 'c', datasize);	
        iter++;
        if(hasToSynch) hipStreamSynchronize(s);
    }

    hipMemcpyAsync(hData,dData,sizeof(float),hipMemcpyDeviceToHost, s);
    hipStreamSynchronize(s);
    
    hipHostFree(hData);
    hipFree(dData);
}
