#include <hip/hip_runtime.h>
#include <malloc.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h> 
#include <math.h>
#include <unistd.h>

//no boundary checks to avoid unnecessary "if"s.
__global__ void copyKernelGPU(float *a, float *b){

    const uint32_t gid = threadIdx.x + blockIdx.x * blockDim.x; 
    b[gid] = a[gid];
}

void printHelp(){
    printf("Usage\n");
    printf("cudameasure [num of float per buffer] [iterations] (dryrun)\n");
    printf("(dryrun): type 'dryrun' as the last arg. to run an unmeasured kernel launch before the actual measure\n");
}

int main(int argc, char *argv[]){

    bool dryrun = false;
    size_t elements = 2048;
    size_t iterations = 1;

    if(argc<3){

        printHelp();
        printf("Not enough arguments. Defaulting to no dry run.\n");
        printf("Num elements = %zu\n", elements);
        printf("Iterations = %zu\n", iterations);

    }else{

        elements = atoi(argv[1]);
        iterations = atoi(argv[2]);
        
        if(argc>3) 
            if(strcmp("dryrun",argv[3])==0)
                dryrun = true;

        printf("Executing with the following parameters: \n");
        printf("Num elements = %zu\n", elements);
        printf("Will perform a dry run? %s\n", (dryrun) ? "yes" : "no" );

    }

    fflush(stdout); 

    const size_t datasize = sizeof(float) * elements;

    hipStream_t s;
    hipStreamCreate(&s);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    float *hData; 
    float *dData0; 
    float *dData1;

    hipHostMalloc((void**)&hData, datasize, hipHostMallocDefault);
    hipMalloc((void**)&dData0, datasize);
    hipMalloc((void**)&dData1, datasize); 
    memset(hData,101,datasize);
    hipMemcpyAsync(dData0, hData, datasize, hipMemcpyHostToDevice, s);
    hipStreamSynchronize(s);

    const uint32_t threads = 128; //TODO: find best.
    const uint32_t blocks = datasize/sizeof(float)/threads; //TODO: check for remainder

    if(dryrun){
        copyKernelGPU<<<blocks,threads,0,s>>>(dData0,dData1);
        hipMemcpyAsync(hData,dData1,datasize,hipMemcpyHostToDevice,s);
        hipStreamSynchronize(s);
        memset(hData,101,datasize);
    }

    usleep(10000);

    for(size_t i=0; i<iterations; i++){
        hipEventRecord(start, s);
        copyKernelGPU<<<blocks,threads,0,s>>>(dData0,dData1);
        hipStreamSynchronize(s);
        hipEventRecord(stop, s);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        printf("%f\n", elapsedTime);
        fflush(stdout);
    }

    hipMemcpyAsync(hData,dData1,datasize,hipMemcpyHostToDevice,s); 
    hipStreamSynchronize(s);

    printf("CCHECK %f\n", hData[0]); 
    fflush(stdout);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(dData0);
    hipFree(dData1);
    hipHostFree(hData);
    hipStreamDestroy(s);

    return EXIT_SUCCESS;
}
